#include "hip/hip_runtime.h"
#include "d_render.h"

__global__ void depth_pcloud_render_kernel(float* pcloud, int* zbuf, int batch_size, int n, int h, int w){
    int pid = blockIdx.x;
    int bid = blockIdx.y;
    float* p_data = pcloud + (bid*n*3 + pid*3);
    int x = max(0, min((int)(w*((*p_data)*0.5 + 0.5)), w-1));
    int y = max(0, min((int)(h*((*(p_data+1))*0.5 + 0.5)), h-1));
    for(int tx = max(0, x-1); tx < min(x+1, w); tx++){   
        for(int ty = max(0, y-1); ty < min(y+1, w); ty++){ 
            atomicMax(zbuf + (bid*h*w + w*y + x), (int)(*(p_data+2)*1e5));
        }
    }
}


void depth_pcloud_render_cuda(float* pcloud, int* zbuf, int batch_size, int n, int h, int w){
    // pcloud B N 3
    // zbuf B H W
    dim3 blocks_dim(n, batch_size);

    depth_pcloud_render_kernel<<<blocks_dim, 1>>>(
        pcloud, zbuf, batch_size, n, h, w
    );
}


__global__ void depth_pcloud_render_idx_kernel(float* pcloud, int* zbuf, int* idbuf, int batch_size, int n, int h, int w){
    int pid = blockIdx.x;
    int bid = blockIdx.y;
    float* p_data = pcloud + (bid*n*3 + pid*3);
    int x = max(0, min((int)(w*((*p_data)*0.5 + 0.5)), w-1));
    int y = max(0, min((int)(h*((*(p_data+1))*0.5 + 0.5)), h-1));
    for(int tx = max(0, x-1); tx < min(x+1, w); tx++){   
        for(int ty = max(0, y-1); ty < min(y+1, w); ty++){ 
            if((int)(*(p_data+2)*1e5) >= *(zbuf + (bid*h*w + w*y + x))){
                atomicExch(idbuf + (bid*h*w + w*y + x), bid*n + pid);
            }
        }
    }
}


void depth_pcloud_render_idx_cuda(float* pcloud, int* zbuf, int* idbuf, int batch_size, int n, int h, int w){
    // pcloud B N 3
    // zbuf B H W
    dim3 blocks_dim(n, batch_size);

    depth_pcloud_render_kernel<<<blocks_dim, 1>>>(
        pcloud, zbuf, batch_size, n, h, w
    );
    depth_pcloud_render_idx_kernel<<<blocks_dim, 1>>>(
        pcloud, zbuf, idbuf, batch_size, n, h, w
    );
}


__global__ void depth_pcloud_render_idx_backward_kernel(float* pcloud, int* idbuf, float* grad, float* grad_out, int batch_size, int n, int h, int w){
    int bid = blockIdx.x;
    int hid = blockIdx.y;
    int wid = blockIdx.z;
    int id_data = *(pcloud + (bid*h*w + hid*w + wid));
    if(id_data >= 0){
        atomicExch(grad_out + id_data, *(grad +  (bid*h*w + hid*w + wid)));
    }
}

void depth_pcloud_render_idx_backward_cuda(float* pcloud, int* idbuf, float* grad, float* grad_out, int batch_size, int n, int h, int w){
    dim3 blocks_dim(batch_size, h, w);

    depth_pcloud_render_idx_backward_kernel<<<blocks_dim, 1>>>(
        pcloud, idbuf, grad, grad_out, batch_size, n, h, w
    );
}